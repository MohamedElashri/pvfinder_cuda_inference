#include "layers/deconv.hpp"

namespace pvfinder {

DeconvLayer::DeconvLayer(int in_ch, int out_ch, int kernel) 
    : in_channels(in_ch), out_channels(out_ch), kernel_size(kernel) {
    
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&biasDesc));
    
    // Set up convolution descriptor for transposed convolution
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(convDesc,
        0, 0,           // padding
        2, 1,           // stride (2 for upsampling)
        1, 1,           // dilation
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT));
    
    // For transposed convolution, we swap in/out channels in filter
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filterDesc,
        HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW,
        in_channels, out_channels,  // swapped for transposed conv
        kernel_size, 1));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(biasDesc,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        1, out_channels, 1, 1));
    
    // Allocate weights and bias
    size_t weights_size = in_channels * out_channels * kernel_size * sizeof(float);
    size_t bias_size = out_channels * sizeof(float);
    
    CUDA_CHECK(hipMalloc(&weights, weights_size));
    CUDA_CHECK(hipMalloc(&bias, bias_size));
}

DeconvLayer::~DeconvLayer() {
    if (weights) CUDA_CHECK(hipFree(weights));
    if (bias) CUDA_CHECK(hipFree(bias));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(convDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(biasDesc));
}

void DeconvLayer::loadWeights(const float* w, const float* b) {
    size_t weights_size = in_channels * out_channels * kernel_size * sizeof(float);
    size_t bias_size = out_channels * sizeof(float);
    
    CUDA_CHECK(hipMemcpy(weights, w, weights_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bias, b, bias_size, hipMemcpyHostToDevice));
}

void DeconvLayer::forward(const Tensor& input, Tensor& output) {
    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&inputDesc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(inputDesc,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        input.dims[0], input.dims[1], input.dims[2], input.dims[3]));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(outputDesc,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        output.dims[0], output.dims[1], output.dims[2], output.dims[3]));
    
    size_t workspace_size = 0;
    CUDNN_CHECK(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnnHandle,
        filterDesc,
        inputDesc,
        convDesc,
        outputDesc,
        HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
        &workspace_size));
    
    void* workspace = nullptr;
    if (workspace_size > 0) {
        CUDA_CHECK(hipMalloc(&workspace, workspace_size));
    }
    
    float alpha = 1.0f, beta = 0.0f;
    
    // Transposed convolution using backward data algorithm
    CUDNN_CHECK(hipdnnConvolutionBackwardData(cudnnHandle,
        &alpha,
        filterDesc,
        weights,
        inputDesc,
        input.data,
        convDesc,
        HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
        workspace,
        workspace_size,
        &beta,
        outputDesc,
        output.data));
    
    // Add bias
    CUDNN_CHECK(hipdnnAddTensor(cudnnHandle,
        &alpha,
        biasDesc,
        bias,
        &alpha,
        outputDesc,
        output.data));
    
    if (workspace) CUDA_CHECK(hipFree(workspace));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(inputDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
}

} // namespace pvfinder