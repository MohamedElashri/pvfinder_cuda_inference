#include "layers/pooling.hpp"

namespace pvfinder {

MaxPoolLayer::MaxPoolLayer(int kernel) : kernel_size(kernel) {
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc));
    
    // Note: For 1D pooling, we use width=1 for other dimensions
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(
        poolingDesc,
        HIPDNN_POOLING_MAX,          // mode
        HIPDNN_NOT_PROPAGATE_NAN,    // nanPropagation
        1,              // window height (1 for 1D)
        kernel_size,    // window width (our actual pooling size)
        0,              // vertical padding
        0,              // horizontal padding
        1,              // vertical stride
        kernel_size     // horizontal stride (same as window width)
    ));
}

MaxPoolLayer::~MaxPoolLayer() {
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(poolingDesc));
}

void MaxPoolLayer::forward(const Tensor& input, Tensor& output) {
    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&inputDesc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
    
    // For 1D data, treat it as 2D with width=1
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
        inputDesc,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        input.dims[0],    // batch size
        input.dims[1],    // channels
        1,                // height (1 for 1D)
        input.dims[2]     // width (our actual data size)
    ));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
        outputDesc,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        output.dims[0],   // batch size
        output.dims[1],   // channels
        1,                // height (1 for 1D)
        output.dims[2]    // width (our actual data size)
    ));
    
    float alpha = 1.0f, beta = 0.0f;
    
    CUDNN_CHECK(hipdnnPoolingForward(cudnnHandle,
        poolingDesc,
        &alpha,
        inputDesc,
        input.data,
        &beta,
        outputDesc,
        output.data));
    
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(inputDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
}

} // namespace pvfinder